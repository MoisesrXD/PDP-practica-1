//---------------------------------
// PROGRAMA BASICO EN C 
//--------------------------------
// Moises Roman Delgadillo Perez
// 27/08/2021
//--------------------------------
//Input and output libraries 

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
//Predefinitons
#define N 512 
#define Real double
//--------------------------------
//Array addition function
//c = a + b
//--------------------------------
void host_add(int* a, int* b, int* c) {
	for (int idx = 0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}
//--------------------------------
//Array filling
//--------------------------------
void fill_array(int* data) {
	for (int idx = 0; idx < N; idx++)
		data[idx] = idx;
}
//--------------------------------
//Data output 
//--------------------------------
void print_output(int* a, int* b, int* c) {
	for (int idx = 0; idx < N; idx++)
	printf("\n %d + %d = %d", a[idx], b[idx], c[idx]);
	printf("\n");
}
//--------------------------------
//Main function
//--------------------------------
int main(void) {
	//Array integers
	int* a, * b, * c;
	//Array memory size
	int size = N * sizeof(int);
	//request and memory filling for arrays
	a = (int*)malloc(size); fill_array(a);
	b = (int*)malloc(size); fill_array(b);
	c = (int*)malloc(size);
	//Calling function addition
	host_add(a, b, c);
	//Writing result in screen 
	print_output(a, b, c);
	//Memory release
	free(a); free(b); free(c);
	//Program success
	return 0;
}
